// RUN: %clang_cc1 -fsycl-is-host -emit-llvm %s -o - | FileCheck %s -check-prefix CHECK-HOST
// RUN: %clang_cc1 -fsycl-is-device -emit-llvm %s -o - | FileCheck %s -check-prefix CHECK-DEV

// Test if a dummy __host__ function (returning undef) is generated for every __device__ function without a host counterpart in sycl-host compilation.

#include "../CodeGenCUDA/Inputs/hip/hip_runtime.h"
#include "Inputs/sycl.hpp"

// CHECK-HOST: ret i32 2
// CHECK-DEV: ret i32 1
__device__ int fun0() { return 1; }
__host__ int fun0() { return 2; }

// CHECK-HOST: ret i32 3
// CHECK-DEV: ret i32 3
__host__ __device__ int fun1() { return 3; }

// CHECK-HOST: ret i32 4
// CHECK-DEV: ret i32 4
__host__ int fun2() { return 4; }

// CHECK-HOST: ret i32 undef
// CHECK-DEV: ret i32 5
__device__ int fun3() { return 5; }

int main(){

  sycl::queue deviceQueue;

  deviceQueue.submit([&](sycl::handler &h) {
    h.single_task<class kern>([]() {
      fun0();
      fun1();
      fun2();
      fun3();
    });
  });

  return 0;
}

